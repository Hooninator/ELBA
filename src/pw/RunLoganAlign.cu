/* Created by Giulia Guidi on 4/16/2021. */

#include "../../src/pw/GPULoganAligner.cpp"
#include "../../include/cuda/logan.cuh"

#define BATCH_SIZE 100000
#define MIN_OV_LEN 10000

void 
RunLoganAlign(vector<string>& seqHs, vector<string>& seqVs, vector<LSeed>& seeds, vector<loganResult>& xscores)
{
	ScoringSchemeL sscheme(1, -1, -1, -1);
	std::vector<ScoringSchemeL> scoring;
	scoring.push_back(sscheme);

	int deviceCount;
    hipGetDeviceCount(&deviceCount);
    omp_set_num_threads(deviceCount); // one OMP thread per GPU

	int AlignmentsToBePerformed = seeds.size();
	int numAlignmentsLocal = BATCH_SIZE * deviceCount; 

	//	Load balancer that divides the work in batches of 100K alignments
	for(int i = 0; i < AlignmentsToBePerformed; i += BATCH_SIZE * deviceCount)
	{
		if(AlignmentsToBePerformed < (i + BATCH_SIZE * deviceCount))
			numAlignmentsLocal = AlignmentsToBePerformed % (BATCH_SIZE * deviceCount);

		int* res = (int*)malloc(numAlignmentsLocal * sizeof(int));	

		std::vector<string>::const_iterator first_t = seqHs.begin() + i;
		std::vector<string>::const_iterator last_t  = seqHs.begin() + i + numAlignmentsLocal;
		std::vector<string> bseqHs(first_t, last_t);

		std::vector<string>::const_iterator first_q = seqVs.begin() + i;
		std::vector<string>::const_iterator last_q  = seqVs.begin() + i + numAlignmentsLocal;
		std::vector<string> bseqVs(first_q, last_q);

		std::vector<LSeed>::const_iterator first_s = seeds.begin() + i;
		std::vector<LSeed>::const_iterator last_s  = seeds.begin() + i + numAlignmentsLocal;
		std::vector<LSeed> bseeds(first_s, last_s);

		extendSeedL(bseeds, EXTEND_BOTHL, bseqHs, bseqVs, scoring, xdrop, seed_length, res, numAlignmentsLocal, deviceCount);

		for(int j = 0; j < numAlignmentsLocal; j++)
		{
			xscores[j+i].score = res[j];
			xscores[j+i].seed  = bseeds[j];
		}

		free(res);
	}
}